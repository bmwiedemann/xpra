/*
 * This file is part of Xpra.
 * Copyright (C) 2013 Antoine Martin <antoine@devloop.org.uk>
 * Xpra is released under the terms of the GNU GPL v2, or, at your option, any
 * later version. See the file COPYING for details.
 */


#include <hip/hip_runtime.h>
#include <stdint.h>

extern "C" __global__ void BGRA_to_U(uint8_t *srcImage, int src_w, int src_h, int srcPitch,
                       uint8_t *dstImage, int dst_w, int dst_h, int dstPitch,
                       int w, int h)
{
    uint32_t gx, gy;
    gx = blockIdx.x * blockDim.x + threadIdx.x;
    gy = blockIdx.y * blockDim.y + threadIdx.y;

    uint32_t src_y = gy * src_h / dst_h;
    uint32_t src_x = gx * src_w / dst_w;

    if ((src_x < w) & (src_y < h)) {
        uint8_t R;
        uint8_t G;
        uint8_t B;
        //one 32-bit RGB pixel at a time:
        uint32_t si = (src_y * srcPitch) + src_x * 4;
        R = srcImage[si+2];
        G = srcImage[si+1];
        B = srcImage[si];

        dstImage[(gy * dstPitch) + gx] = __float2int_rn(-0.148 * R - 0.291 * G + 0.439 * B + 128);
    }
}
