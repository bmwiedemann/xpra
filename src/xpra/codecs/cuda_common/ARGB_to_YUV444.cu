/*
 * This file is part of Xpra.
 * Copyright (C) 2013-2017 Antoine Martin <antoine@devloop.org.uk>
 * Xpra is released under the terms of the GNU GPL v2, or, at your option, any
 * later version. See the file COPYING for details.
 */


#include <hip/hip_runtime.h>
#include <stdint.h>

extern "C" __global__ void ARGB_to_YUV444(uint8_t *srcImage, int src_w, int src_h, int srcPitch,
                             uint8_t *dstImage, int dst_w, int dst_h, int dstPitch,
                             int w, int h)
{
    uint32_t gx, gy;
    gx = blockIdx.x * blockDim.x + threadIdx.x;
    gy = blockIdx.y * blockDim.y + threadIdx.y;

    uint32_t src_y = gy * src_h / dst_h;
    uint32_t src_x = gx * src_w / dst_w;

    if ((src_x < w) & (src_y < h)) {
        uint8_t R;
        uint8_t G;
        uint8_t B;
        //one 32-bit RGB pixel at a time:
        uint32_t si = (src_y * srcPitch) + src_x * 4;
        R = srcImage[si+1];
        G = srcImage[si+2];
        B = srcImage[si+3];

        uint32_t di;
        di = (gy * dstPitch) + gx;
        dstImage[di] = __float2int_rn(0.257 * R + 0.504 * G + 0.098 * B + 16);
        di += dstPitch*dst_h;
        dstImage[di] = __float2int_rn(-0.148 * R - 0.291 * G + 0.439 * B + 128);
        di += dstPitch*dst_h;
        dstImage[di] = __float2int_rn(0.439 * R - 0.368 * G - 0.071 * B + 128);
    }
}
