/*
 * This file is part of Xpra.
 * Copyright (C) 2013 Antoine Martin <antoine@devloop.org.uk>
 * Xpra is released under the terms of the GNU GPL v2, or, at your option, any
 * later version. See the file COPYING for details.
 */


#include <hip/hip_runtime.h>
#include <stdint.h>

extern "C" __global__ void BGRA_to_NV12(uint8_t *srcImage, int src_w, int src_h, int srcPitch,
                          uint8_t *dstImage, int dst_w, int dst_h, int dstPitch,
                          int w, int h)
{
    uint32_t gx, gy;
    gx = blockIdx.x * blockDim.x + threadIdx.x;
    gy = blockIdx.y * blockDim.y + threadIdx.y;

    uint32_t src_y = gy*2 * src_h / dst_h;
    uint32_t src_x = gx*2 * src_w / dst_w;

    if ((src_x < w) & (src_y < h)) {
        //4 bytes per pixel, and 2 pixels width/height at a time:
        //byte index:
        uint32_t si = (src_y * srcPitch) + src_x * 4;

        //we may read up to 4 32-bit RGB pixels:
        uint8_t R[4];
        uint8_t G[4];
        uint8_t B[4];
        uint8_t j = 0;
        R[0] = srcImage[si+2];
        G[0] = srcImage[si+1];
        B[0] = srcImage[si];
        for (j=1; j<4; j++) {
            R[j] = R[0];
            G[j] = G[0];
            B[j] = B[0];
        }

        //write up to 4 Y pixels:
        uint32_t di = (gy * 2 * dstPitch) + gx * 2;
        dstImage[di] = __float2int_rn(0.257 * R[0] + 0.504 * G[0] + 0.098 * B[0] + 16);
        if (gx*2 + 1 < src_w) {
            R[1] = srcImage[si+6];
            G[1] = srcImage[si+5];
            B[1] = srcImage[si+4];
            dstImage[di + 1] = __float2int_rn(0.257 * R[1] + 0.504 * G[1] + 0.098 * B[1] + 16);
        }
        if (gy*2 + 1 < src_h) {
            si += srcPitch;
            di += dstPitch;
            R[2] = srcImage[si+2];
            G[2] = srcImage[si+1];
            B[2] = srcImage[si];
            dstImage[di] = __float2int_rn(0.257 * R[2] + 0.504 * G[2] + 0.098 * B[2] + 16);
            if (gx*2 + 1 < src_w) {
                R[3] = srcImage[si+6];
                G[3] = srcImage[si+5];
                B[3] = srcImage[si+4];
                dstImage[di + 1] = __float2int_rn(0.257 * R[3] + 0.504 * G[3] + 0.098 * B[3] + 16);
            }
        }

        //write 1 U and 1 V pixel:
        float u = 0;
        float v = 0;
        for (j=0; j<4; j++) {
            u += -0.148 * R[j] - 0.291 * G[j] + 0.439 * B[j] + 128;
            v +=  0.439 * R[j] - 0.368 * G[j] - 0.071 * B[j] + 128;
        }
        di = (dst_h + gy) * dstPitch + gx * 2;
        dstImage[di]      = __float2int_rn(u / 4.0);
        dstImage[di + 1]  = __float2int_rn(v / 4.0);
    }
}
