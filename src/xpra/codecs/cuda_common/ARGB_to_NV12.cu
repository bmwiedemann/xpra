/*
 * This file is part of Xpra.
 * Copyright (C) 2013-2017 Antoine Martin <antoine@devloop.org.uk>
 * Xpra is released under the terms of the GNU GPL v2, or, at your option, any
 * later version. See the file COPYING for details.
 */


#include <hip/hip_runtime.h>
#include <stdint.h>

extern "C" __global__ void ARGB_to_NV12(uint8_t *srcImage, int src_w, int src_h, int srcPitch,
                          uint8_t *dstImage, int dst_w, int dst_h, int dstPitch,
                          int w, int h)
{
    uint32_t gx, gy;
    gx = blockIdx.x * blockDim.x + threadIdx.x;
    gy = blockIdx.y * blockDim.y + threadIdx.y;

    uint32_t src_y = gy*2 * src_h / dst_h;
    uint32_t src_x = gx*2 * src_w / dst_w;

    if ((src_x < w) & (src_y < h)) {
        //4 bytes per pixel, and 2 pixels width/height at a time:
        //byte index:
        uint32_t si = (src_y * srcPitch) + src_x * 4;

        //we may read up to 4 32-bit RGB pixels:
        uint8_t R[4];
        uint8_t G[4];
        uint8_t B[4];
        uint8_t j = 0;
        R[0] = srcImage[si+1];
        G[0] = srcImage[si+2];
        B[0] = srcImage[si+3];
        for (j=1; j<4; j++) {
            R[j] = R[0];
            G[j] = G[0];
            B[j] = B[0];
        }

        //write up to 4 Y pixels:
        uint32_t di = (gy * 2 * dstPitch) + gx * 2;
        dstImage[di] = __float2int_rn(0.257 * R[0] + 0.504 * G[0] + 0.098 * B[0] + 16);
        if (gx*2 + 1 < src_w) {
            R[1] = srcImage[si+5];
            G[1] = srcImage[si+6];
            B[1] = srcImage[si+7];
            dstImage[di + 1] = __float2int_rn(0.257 * R[1] + 0.504 * G[1] + 0.098 * B[1] + 16);
        }
        if (gy*2 + 1 < src_h) {
            si += srcPitch;
            di += dstPitch;
            R[2] = srcImage[si+1];
            G[2] = srcImage[si+2];
            B[2] = srcImage[si+3];
            dstImage[di] = __float2int_rn(0.257 * R[2] + 0.504 * G[2] + 0.098 * B[2] + 16);
            if (gx*2 + 1 < src_w) {
                R[3] = srcImage[si+5];
                G[3] = srcImage[si+6];
                B[3] = srcImage[si+7];
                dstImage[di + 1] = __float2int_rn(0.257 * R[3] + 0.504 * G[3] + 0.098 * B[3] + 16);
            }
        }

        //write 1 U and 1 V pixel:
        float u = 0;
        float v = 0;
        for (j=0; j<4; j++) {
            u += -0.148 * R[j] - 0.291 * G[j] + 0.439 * B[j] + 128;
            v +=  0.439 * R[j] - 0.368 * G[j] - 0.071 * B[j] + 128;
        }
        di = (dst_h + gy) * dstPitch + gx * 2;
        dstImage[di]      = __float2int_rn(u / 4.0);
        dstImage[di + 1]  = __float2int_rn(v / 4.0);
    }
}
